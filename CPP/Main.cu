#include "hip/hip_runtime.h"
#include "pch.h"

#include <chrono>
#include <iostream>
#include <vector>

#include "env.hpp"
#include "Packer.cuh"
#include "Raster.hpp"
#include "RasterUtils.hpp"
#include "CudaMemManager.cuh"

// some common constants

// number of vertical rounds per block
//  in kernels that suppourt vertical segmentation
constexpr unsigned int block_height_rounds = 32;

// standard block width in number of threads
constexpr unsigned int std_block_width = 32;

// expanded block width in number of threads
constexpr unsigned int expanded_block_with = 256;

inline unsigned int number_vertical_blocks(const int n_vertical_items){
    static_assert(block_height_rounds == 32);
    return CEIL_DIV_32(n_vertical_items);
}

inline unsigned int number_horizontal_blocks(const int n_horizontal_items){
    static_assert(std_block_width == 32);
    return CEIL_DIV_32(n_horizontal_items);
}

inline unsigned int expanded_number_horizontal_block(const int n_horizontal_items){
    static_assert(expanded_block_with == 256);
    return CEIL_DIV_256(n_horizontal_items);
}



// display certain device statistics to the console
void displayCUDAdeviceStats(void){
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    std::cout << "Detected " << deviceCount << " devices" << std::endl;

    if(deviceCount <= 0){
        std::cout << "Error no device detected" << std::endl;
        std::exit(2);
    }

    for (int device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        printf("Device %d has compute capability %d.%d.\n",
            device, deviceProp.major, deviceProp.minor);
        printf("  %s\n", deviceProp.name);
        printf("  %dkHz, %d threads per block, %d threads per mp, %d mp on device, %d warp size\n",
            deviceProp.clockRate, deviceProp.maxThreadsPerBlock, 
            deviceProp.maxThreadsPerMultiProcessor,
            deviceProp.multiProcessorCount, deviceProp.warpSize);
        // printf("  %d i32 reg per block, %d i32 reg per mp\n", deviceProp.regsPerBlock, deviceProp.regsPerMultiprocessor);

        printf("  %lu MB device global mem, %lu (%lu opt) B shared mem per block\n", 
            (deviceProp.totalGlobalMem) >> 20, 
            (deviceProp.sharedMemPerBlock),
            (deviceProp.sharedMemPerBlockOptin));

        if(deviceProp.kernelExecTimeoutEnabled){
            printf("  [CRITICAL] Kernel Timeout is enabled (no idea what it is lol)\n");
        }
    }
}


__device__ inline int fromXY(const unsigned int x, const unsigned int y, const unsigned int width_stride){
    return y * width_stride + x;
}

template <int n_rounds>
__global__ void calculateCollisions(
    uint32_t* sheet_ptr, const unsigned int sheet_pitch_uint_32,
    uint32_t* output_ptr, const unsigned int output_pitch_uint_32,
    uint32_t* part_prt, const unsigned int part_pitch_uint32,
    const unsigned int sheet_width, const unsigned int sheet_height,
    const unsigned int part_width, const unsigned int part_height)
{
#ifdef DEBUG
    if(blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0){
        printf(
            "============== Calculate Collisions Entry ===================\n"
            "%p %p %p\n"
            "%u %u %u\n"
            "%u %u %u %u\n"
            "{%u %u %u} {%u %u %u}\n"
            "=============================================================\n",
            sheet_ptr, output_ptr, part_prt,
            sheet_pitch_uint_32, output_pitch_uint_32, part_pitch_uint32,
            sheet_width, sheet_height, part_width, part_height,
            gridDim.x, gridDim.y, gridDim.z,
            blockDim.x, blockDim.y, blockDim.z
        );
    }
#endif

    // if(threadIdx.x == 0){
    //     printf("Hello from block %d %d, thread %d\n", blockIdx.x, blockIdx.y, threadIdx.x);
    // }

    // the x_block coordinate we are responsible for
    unsigned int my_x = blockIdx.x*blockDim.x + threadIdx.x;

    if(my_x > sheet_width){
        __syncthreads();
        return;
    }

    for(unsigned int round_y_offset = 0; round_y_offset < n_rounds; round_y_offset++){
        // miniumum possible region of the part that can overlap

        // the y_block coordinate we are responsible for
        unsigned int my_y = blockIdx.y*blockDim.y*n_rounds + round_y_offset;

        // the value of the sheet at the given cell
        const uint32_t sheet_value = sheet_ptr[fromXY(my_x, my_y, sheet_pitch_uint_32)];

        // TODO - remove
        //  sheet ptr should be const; this is just for testing
        // sheet_ptr[fromXY(my_x, my_y, sheet_pitch_uint_32)] = ~0;

        // TODO - check if these are really what we want
        const unsigned int min_part_x = min(my_x, part_width);
        const unsigned int min_part_y = min(my_y, part_height);
        const unsigned int max_part_x = ((part_width + my_x > sheet_width) ? (part_width + my_x - sheet_width) : 0);
        const unsigned int max_part_y = ((part_height + my_y > sheet_height) ? (part_height + my_y - sheet_height) : 0);

        // TODO - actually figure out
        for(unsigned int current_part_x = min_part_x; current_part_x <= max_part_x; current_part_x++){ // over x
            for(unsigned int current_part_y=min_part_y; current_part_y <= max_part_y; current_part_y++){ // over y

                // TODO - no need to refetch a2 from global mem, it should be old a1
                const uint32_t a1 = sheet_ptr[fromXY(current_part_x, current_part_y, part_pitch_uint32)];
                const uint32_t a2 = ((current_part_y) ? (sheet_ptr[fromXY(current_part_x, current_part_y-1, part_pitch_uint32)]) : (0));

                uint32_t c = 0;

                #pragma unroll
                for(unsigned int i = 0; i < 32; i++){
                    const uint32_t t = ((a1 << i) & sheet_value) | ((a2 >> (32-i)) & sheet_value);
                    c |= (t ? (1 << i) : 0);
                }

                atomicOr(output_ptr + fromXY(current_part_x + my_x, current_part_y + my_y, output_pitch_uint_32), c);
            }
        }
    }

    __syncthreads();
}

// for each x_value find the lowest non-colliding y value
//  i.e. the first zero bit
__global__ void findBestPlacement(
    uint32_t const * const output_ptr, const unsigned int output_pitch_uint_32,
    const unsigned output_width, const unsigned int output_height_reg_32,
    uint32_t * storage
){
#ifdef DEBUG
    if(blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0){
        printf(
            "============== Find Best Placement Entry ====================\n"
            "%p %p\n"
            "%u %u %u\n"
            "{%u %u %u} {%u %u %u}\n"
            "=============================================================\n",
            output_ptr, storage,
            output_pitch_uint_32, output_width, output_height_reg_32,
            gridDim.x, gridDim.y, gridDim.z,
            blockDim.x, blockDim.y, blockDim.z
        );
    }
#endif

    // x-value of this worker
    const unsigned int my_x = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ uint32_t s[];

    // initalize shared memory
    s[threadIdx.x*2] = my_x;
    s[threadIdx.x*2+1] = ~0;

    if(my_x >= output_width){
        // pass
    }else{
        unsigned int best_y = output_height_reg_32;
        for(unsigned int y_offset = 0; y_offset < output_height_reg_32; y_offset++){
            const unsigned int c = output_ptr[fromXY(my_x, y_offset, output_pitch_uint_32)];

            if(c == ~0){
                // all of the spots are collisions 
                continue;
            }

            constexpr uint8_t lut[] = {0,1,0,2,0,1,0,3,0,1,0,2,0,1,0,4};

            // there is a zero somewhere, so we can find it easily
            best_y = 31;
            #pragma unroll
            for(unsigned int i = 0; i < 32; i+=4){
                const uint32_t t = lut[(c & (0xF << i)) >> i];

                if(t == 4){
                    continue;
                }

                best_y = t + i;
                break;
            }
            
            s[threadIdx.x*2+1] = best_y + (y_offset * 32);
            break;
        }
    }
    __syncthreads();

    #pragma unroll
    for(unsigned int i = 1; i <= 8; i++){
        if((threadIdx.x & ((1 << i) - 1)) == 0){
            // when i = 1 take even threads
            // when i = 2 take every 4th thread
            // when i = 3 take every 8th thread

            // reduce, taking the left most item
            if(s[threadIdx.x*2+1] == ~0){
                // we did not find any valid position at this x value
                // so take the other position
                //  if its also invalid thats fine
                //  strictly speaking, its not worse
                s[threadIdx.x*2] = s[(threadIdx.x+(1 << (i - 1)))*2];
                s[threadIdx.x*2+1] = s[(threadIdx.x+(1 << (i - 1)))*2+1];
            }
        }
        __syncthreads();
    }

    if(threadIdx.x == 0){
        storage[blockIdx.x*2] = s[0];
        storage[blockIdx.x*2+1] = s[1];
    }
}


// bake a part into the sheet at a given x,y location
// TODO - error in here somewhere when y != 0
template <int n_rounds>
__global__ void bakePart(
    uint32_t* part_ptr, const unsigned int part_pitch_uint32,
    uint32_t* sheet_ptr, const unsigned int sheet_pitch_uint32,
    const unsigned int x, const unsigned int y,
    const unsigned int part_width, const unsigned int part_height
){
    const unsigned int base_x = CLAMP_32(x);
    const unsigned int base_y = CLAMP_32(y);
    const unsigned int part_height_r32 = CEIL_DIV_32(part_height);

    return;
    // the x_block coordinate we are responsible for
    //  relative to sheet coordinates
    const unsigned int sheet_x = blockIdx.x*blockDim.x + threadIdx.x + base_x;
    const unsigned int part_x = sheet_x - x; // may underflow and thats ok

    if(part_x >= part_width){
        // we are either left or right outside
        //  and thus have nothing to do
        __syncthreads();
        return;
    }

    // the above may alternatively be expressed as
    // if(sheet_x < x){
    //     // this thread is left of the part, and thus nothing to do
    //     __syncthreads();
    //     return;
    // }
    // if(sheet_x >= x + part_width){
    //     // this thread is right of the part, and thus nothing to do
    //     __syncthreads();
    //     return;
    // }

    uint32_t part_lower = 0;
    uint32_t part_upper = 0;

    const auto shift_amt = FAST_MOD_32(y);
    const auto shift_amt_comp = (32 - y);
    
    for(unsigned int round_y_offset = 0; round_y_offset < n_rounds; round_y_offset++){
        // the y_block coordinate we are responsible for
        //  relative to the sheet
        unsigned int sheet_y = blockIdx.y*blockDim.y*n_rounds + round_y_offset + base_y;

        unsigned int part_y = sheet_y - y; // this may underflow and thats a good thing

        // >= is critical for checking alignment
        if(part_y >= part_height_r32){
            // either underflowed so need another round
            // or overflowed and could break
            //  but its not that slow
            // alternatively: could explicitly check the two conditions
            continue;
        }

        part_lower = part_upper; // shift down previous
        part_upper = ((part_y < part_height_r32) ? part_ptr[fromXY(part_x, part_y, part_pitch_uint32)] : 0); // fetch new

        const uint32_t c = ((shift_amt == 0) ? (part_upper) : ((part_upper << shift_amt) & (part_lower >> shift_amt_comp)));

        const auto t = atomicOr(sheet_ptr + fromXY(sheet_x, sheet_y, sheet_pitch_uint32), c);

        if (t & c){
            printf("WARNING: Baking part produced a colliding cofiguration\n");
        }
    }

    __syncthreads();
}

void calculateCollisionsWrapper(
    const CudaMemManager2D& sheet, const CudaMemManager2D& part, 
    CudaMemManager2D& collision, CudaMemManager2D& reduce,
    const unsigned int sheet_true_h, const unsigned int sheet_true_w, 
    const unsigned int part_true_h, const unsigned int part_true_w,
    const unsigned int collision_true_h, const unsigned int collision_true_w
){
    assert(sheet.width() == sheet_true_w);
    assert(part.width() == part_true_w);
    assert(collision.width() == collision_true_w);

    printf("Starting a collision calculation\n");
    const dim3 block_shape_c = dim3(std_block_width);
    const dim3 grid_shape_c = dim3(number_horizontal_blocks(sheet.width()), number_vertical_blocks(sheet.height()));

    const dim3 block_shape_r = dim3(expanded_block_with);
    const dim3 grid_shape_r = dim3(expanded_number_horizontal_block(collision.width()));

    const auto sheet_params = sheet.getDeviceParameters();
    const auto part_params = part.getDeviceParameters();
    const auto collision_params = collision.getDeviceParameters();
    const auto reduce_params = reduce.getDeviceParameters();

#ifdef DEBUG
    printf("!> Collision Config: {%u} {%u %u %u} {%u %u %u}\n", 
        block_height_rounds,
        grid_shape_c.x, grid_shape_c.y, grid_shape_c.z,
        block_shape_c.x, block_shape_c.y, block_shape_c.z
    );
    printf("!> Collision Args: <ptr> %lu <ptr> %lu <prt> %lu {%u %u %u %u}\n",
        sheet_params.second, collision_params.second, part_params.second,
        sheet_true_w, sheet_true_h, part_true_w, part_true_h
    );
    printf("!> Reduce Config: {%u %u %u} {%u %u %u} {%lu}\n",
        grid_shape_r.x, grid_shape_r.y, grid_shape_r.z,
        block_shape_r.x, block_shape_r.y, block_shape_r.z,
        sizeof(uint32_t)*2*expanded_block_with
    );
    printf("!> Reduce Args: <ptr> %lu %u %lu <ptr>\n", 
        collision_params.second, collision_true_w, collision.height()
    );
#endif

    const auto k_start_time = std::chrono::high_resolution_clock::now();

    calculateCollisions<block_height_rounds><<<grid_shape_c, block_shape_c>>>(
        sheet_params.first, sheet_params.second,
        collision_params.first, collision_params.second,
        part_params.first, part_params.second,
        sheet_true_w, sheet_true_h,
        part_true_w, part_true_h
    );

    checkCudaError(__LINE__);

    findBestPlacement<<<grid_shape_r, block_shape_r, sizeof(uint32_t)*2*expanded_block_with>>>(
        collision_params.first, collision_params.second,
        collision_true_w, collision.height(),
        reduce_params.first
    );

    CUDACall(hipDeviceSynchronize());

    const auto k_end_time = std::chrono::high_resolution_clock::now();
    
    reduce.Pull();

    printf("Reduce Results: ");
    for(unsigned int i = 0; i < grid_shape_r.x; i++){
        printf("{%u %u} ", reduce.at(i*2,0), reduce.at(i*2+1, 0));
    }
    printf("\n");

    printf("Reduce stats: %lu x %lu (%u) s: %lu b: %lu %p\n", reduce.width(), reduce.height(), grid_shape_r.x,
        reduce.getHostStride(), reduce.getRawWidthBytes(), reduce.gethostPtr_unsafe()
    );

    {
        printf("Old method:\n");
        uint32_t* storage_ptr_uint32 = (uint32_t*)reduce.gethostPtr_unsafe();
        printf("Reduce %u: ", CEIL_DIV_256(collision_true_w));
        for(unsigned int i = 0; i < CEIL_DIV_256(collision_true_w); i++){
            printf("{%u %u}, ", storage_ptr_uint32[i*2], storage_ptr_uint32[i*2+1]);
        }
        printf("\n");
    }

    // TODO - remainder of this 
    //  pull out the answer
    const int NOT_SET = 0b10101;

    const auto end_time = std::chrono::high_resolution_clock::now();

    printf("Collsion calc returned %d %d in %ld us (%ld us kernel)\n",
        NOT_SET, NOT_SET,
        std::chrono::duration_cast<std::chrono::microseconds>(end_time-k_start_time).count(),
        std::chrono::duration_cast<std::chrono::microseconds>(k_end_time-k_start_time).count()
    );

    // void* storage_ptr = malloc(sizeof(uint32_t)*num_reduce_storage_blocks*2);
    //  const uint32_t* storage_ptr_uint32 = (uint32_t*)(storage_ptr);

    // printf("Reduce %u: ", num_reduce_storage_blocks);
    // for(unsigned int i = 0; i < num_reduce_storage_blocks; i++){
    //     printf("{%u %u}, ", storage_ptr_uint32[i*2], storage_ptr_uint32[i*2+1]);
    // }

}

void simple_cuda(const Raster& part){
    // lets compute some constants
    
    // width of the sheet in number of samples
    constexpr size_t sheet_width_samples = SHEET_WIDTH_INCH*SAMPLES_PER_INCH;

    // height of the sheet in number of samples
    constexpr size_t sheet_height_samples = SHEET_HEIGHT_INCH*SAMPLES_PER_INCH;

    // width of the ouput in number of samples
    const size_t output_width_samples = sheet_width_samples - part.getWidth() + 1;

    // height of the ouput in number of samples
    const size_t output_height_samples = sheet_height_samples - part.getHeight() + 1;

    // with of the part in number of samples
    const size_t part_width_samples = part.getWidth();

    // height of the part in number of samples
    const size_t part_height_samples = part.getHeight();



    // i just assume this is true
    //  and not sure what will break if it isnt
    //  next two statements for sure, but what else
    static_assert(sizeof(uint32_t) == 4);

    const bool b_sync = true;

    CudaMemManager2D sheet_mem(sheet_width_samples, CEIL_DIV_32(sheet_height_samples), b_sync);
    CudaMemManager2D part_mem(part, b_sync);
    printf("Collisions: ");
    CudaMemManager2D collisions_mem(output_width_samples, CEIL_DIV_32(output_height_samples), b_sync);
    printf("Reduce: ");
    CudaMemManager2D reduce_mem(expanded_number_horizontal_block(output_width_samples)*2, 1, b_sync);

    CudaMemManager2D::Sync();

    printf("Done initilizing Managed 2D Memory\n");

    calculateCollisionsWrapper(
        sheet_mem, part_mem, collisions_mem, reduce_mem,
        sheet_height_samples, sheet_width_samples, 
        part_height_samples, part_width_samples,
        output_height_samples, output_width_samples
    );

    // return;



    // height of the sheet in number of 32bit registers
    constexpr size_t sheet_height_reg_32 = CEIL_DIV_32(sheet_height_samples);

    // with of a row in memory in bytes
    constexpr size_t row_width_bytes = sizeof(uint32_t)*sheet_width_samples;

    // height of the output in number of 32bit registers
    const size_t output_height_reg_32 = CEIL_DIV_32(output_height_samples);
    
    // width of a row in number of bytes
    const size_t output_row_width_bytes = sizeof(uint32_t)*output_width_samples;


    // height of the part in number of 32bit registers
    const size_t part_height_reg_32 = CEIL_DIV_32(part_height_samples);

    // width of a row in number of bytes
    const size_t part_row_width_bytes = sizeof(uint32_t)*part_width_samples;
    
    // TODO - const qualify, const cast these six

    // pointer to __device__ memory allocated for storing sheet
    void* sheet_devptr;

    // value of the pitch for sheet device pointer
    //  subsequent rows are aligned to bus breaks (512 bytes?)
    size_t sheet_devpitch;

    // pointer to __device__ memory allocated for storing the part
    void* part_devptr = nullptr;

    // value of the pitch for part device pointer
    size_t part_devpitch;

    // pointer to __device__ memort allocated for storing the part
    void* output_devptr;

    // value of the pitch for output device pointer
    size_t output_devpitch;

    CUDACall(hipMallocPitch(&sheet_devptr, &sheet_devpitch, row_width_bytes, sheet_height_reg_32));
    CUDACall(hipMemset2D(sheet_devptr, sheet_devpitch, 0, row_width_bytes, sheet_height_reg_32));
    CUDACall(hipMallocPitch(&output_devptr, &output_devpitch, output_row_width_bytes, output_height_reg_32));
    CUDACall(hipMemset2D(output_devptr, output_devpitch, 0, output_row_width_bytes, output_height_reg_32));
    CUDACall(hipMallocPitch(&part_devptr, &part_devpitch, part_row_width_bytes, part_height_reg_32));
    CUDACall(hipMemset2D(part_devptr, part_devpitch, 0, part_row_width_bytes, part_height_reg_32));
    CUDACall(hipDeviceSynchronize()); // force the prior operations to complete before proceeding

    // create the storage pointer
    const int num_reduce_storage_blocks = CEIL_DIV_256(output_width_samples);

    void* storage_devptr;
    // CUDACall(hipMalloc(&storage_devptr, sizeof(uint32_t)*num_reduce_storage_blocks*2));
    storage_devptr = reduce_mem.getDevPtr_unsafe();

    printf("Allocated arrays successfully.\n");

    // calculate block constants

    // number of threads per block (width)
    constexpr size_t block_width_thread = 32;

    // number of rounds that each thread will perform
    constexpr size_t block_height_rounds = 32;

    // calculate kernel constants

    // number of blocks in the kernel width
    constexpr size_t grid_width_blocks = FAST_DIV_32(sheet_width_samples);
    static_assert(block_width_thread == 32);

    constexpr size_t grid_height_blocks = FAST_DIV_32(sheet_height_reg_32);
    static_assert(block_height_rounds == 32);

    printf(
        "Blocks are %ld threads by %ld rounds, kernel is %ldx%ld blocks (%ldx%ld regs); "
        "Total %ld blocks.\n",
        block_width_thread, block_height_rounds,
        grid_width_blocks, grid_height_blocks, 
        sheet_width_samples, sheet_height_reg_32,
        grid_width_blocks * grid_height_blocks
    );

    //copy in the data for the part
    // {
        const char* c = part.linearPackData();
        const size_t lcl_pitch = part.getLinearPackStride();
        CUDACall(hipMemcpy2D(part_devptr, part_devpitch, c, lcl_pitch, part.getWidth() * sizeof(uint32_t), CEIL_DIV_32(part.getHeight()), hipMemcpyHostToDevice))
    //     delete c;
    // }

    // launch and run the kernel

    const dim3 block_shape = dim3(block_width_thread);
    const dim3 grid_shape = dim3(grid_width_blocks, grid_height_blocks);

#ifdef DEBUG
    printf("!> Collision Config: {%lu} {%u %u %u} {%u %u %u}\n", 
        block_height_rounds,
        grid_shape.x, grid_shape.y, grid_shape.z,
        block_shape.x, block_shape.y, block_shape.z
    );
    printf("!> Collision Args: <ptr> %lu <ptr> %lu <prt> %lu {%lu %lu %u %u}\n",
        sheet_devpitch / sizeof(uint32_t),  output_devpitch / sizeof(uint32_t), part_devpitch / sizeof(uint32_t),
        sheet_width_samples, sheet_height_samples, part.getWidth(), part.getHeight()
    );
    printf("!> Reduce Args: <ptr> %lu %lu %lu <ptr>\n",
        output_devpitch / sizeof(uint32_t),
        output_width_samples, output_height_reg_32
    );
    printf("!> Reduce Config: {%d %u %u} {%u %u %u} {%lu}\n", 
        dim3(num_reduce_storage_blocks).x, dim3(num_reduce_storage_blocks).y, dim3(num_reduce_storage_blocks).z,
        dim3(256).x, dim3(256).y, dim3(256).z,
        256*sizeof(uint32_t)*2
    );
    printf("!> Reduce Args: <ptr> %lu %lu %lu <ptr>\n",
        output_devpitch / sizeof(uint32_t),
        output_width_samples, output_height_reg_32
    );
#endif

    const auto start_time = std::chrono::high_resolution_clock::now();
    calculateCollisions<block_height_rounds><<<grid_shape, block_shape>>>(
        (uint32_t*)sheet_devptr, sheet_devpitch / sizeof(uint32_t),
        (uint32_t*)output_devptr, output_devpitch / sizeof(uint32_t),
        (uint32_t*)part_devptr, part_devpitch / sizeof(uint32_t),
        sheet_width_samples, sheet_height_samples,
        part.getWidth(), part.getHeight()
    );

    checkCudaError(__LINE__);
    
    // CUDACall(hipDeviceSynchronize());
    findBestPlacement<<<num_reduce_storage_blocks, 256, 256*sizeof(uint32_t)*2>>>(
        (uint32_t*)output_devptr, output_devpitch / sizeof(uint32_t),
        output_width_samples, output_height_reg_32,
        (uint32_t*)storage_devptr
    );

    checkCudaError(__LINE__);
    
    CUDACall(hipDeviceSynchronize());
    const auto end_time = std::chrono::high_resolution_clock::now();

    printf("Kernel took %ld us to run\n", 
        std::chrono::duration_cast<std::chrono::microseconds>(end_time-start_time).count()
    );

    // TODO - these should actually be computed
    const unsigned int bake_pos_x = 0;
    // TODO - these should actually be computed
    const unsigned int bake_pos_y = 0;

    static_assert(block_height_rounds == 32);
    static_assert(block_width_thread == 32);

    const unsigned int num_blocks_x = CEIL_DIV_32(bake_pos_x + part.getWidth()) - FAST_DIV_32(bake_pos_x);
    const unsigned int num_blocks_y = CEIL_DIV_32(bake_pos_y + CEIL_DIV_32(part.getHeight())) - FAST_DIV_32(FAST_DIV_32(bake_pos_y));

    printf("Bake kernel is %u x %u blocks\n", num_blocks_x, num_blocks_y);

    // bakePart<block_height_rounds><<<dim3(num_blocks_x, num_blocks_y), dim3(block_width_thread)>>>(
    //     (uint32_t*)part_devptr, part_devpitch,
    //     (uint32_t*)sheet_devptr, sheet_devpitch,
    //     bake_pos_x, bake_pos_y,
    //     part.getWidth(), part.getHeight()
    // );
    // checkCudaError(__LINE__);
    
    CUDACall(hipDeviceSynchronize());
    printf("Finished baking part\n");

    // {
    //     // bakePart<block_height_rounds><<<dim3(num_blocks_x, num_blocks_y), dim3(block_width_thread)>>>(
    //     //     (uint32_t*)part_devptr, part_devpitch,
    //     //     (uint32_t*)sheet_devptr, sheet_devpitch,
    //     //     0, 1024,
    //     //     part.getWidth(), part.getHeight()
    //     // );
    //     calculateCollisions<block_height_rounds><<<grid_shape, block_shape>>>(
    //         (uint32_t*)sheet_devptr, sheet_devpitch / sizeof(uint32_t),
    //         (uint32_t*)output_devptr, output_devpitch / sizeof(uint32_t),
    //         (uint32_t*)part_devptr, part_devpitch / sizeof(uint32_t),
    //         sheet_width_samples, sheet_height_samples,
    //         part.getWidth(), part.getHeight()
    //     );

    //     checkCudaError(__LINE__);
        
    //     // CUDACall(hipDeviceSynchronize());
    //     findBestPlacement<<<num_reduce_storage_blocks, 256, 256*sizeof(uint32_t)*2>>>(
    //         (uint32_t*)output_devptr, output_devpitch / sizeof(uint32_t),
    //         output_width_samples, output_height_reg_32,
    //         (uint32_t*)storage_devptr
    //     );
    // }

    // allocate local memory to store the output

    // number of local bytes to represent the sheet
    constexpr size_t local_sheet_bytes = row_width_bytes * sheet_height_reg_32;

    // number of local bytes to represent the output
    const size_t local_output_bytes = output_row_width_bytes * output_height_reg_32;

    // ptr to local memory for the sheet
    void* sheet_ptr = malloc(local_sheet_bytes);
    memset(sheet_ptr, 0b1010, local_sheet_bytes);
    const char* const sheet_ptr_c = (char*)sheet_ptr;

    // ptr to local memory for the output
    void* output_ptr = malloc(local_output_bytes);
    memset(output_ptr, 0b1010, local_output_bytes);
    const char* const output_ptr_c = (char*)output_ptr;

    // ptr to local memory for the storage
    void* storage_ptr = malloc(sizeof(uint32_t)*num_reduce_storage_blocks*2);
    const uint32_t* storage_ptr_uint32 = (uint32_t*)(storage_ptr);

    assert(collisions_mem.getDeviceStride() == output_devpitch);
    printf("%p %lu (%lu)\n", collisions_mem.getDevPtr_unsafe(), collisions_mem.getDeviceStride(), output_devpitch);

    CUDACall(hipMemcpy2D(sheet_ptr, row_width_bytes, sheet_devptr, sheet_devpitch, row_width_bytes, sheet_height_reg_32, hipMemcpyDeviceToHost));
    CUDACall(hipMemcpy2D(output_ptr, output_row_width_bytes, output_devptr, output_devpitch, output_row_width_bytes, output_height_reg_32, hipMemcpyDeviceToHost));
    //CUDACall(hipMemcpy2D(output_ptr, output_row_width_bytes, collisions_mem.getDevPtr_unsafe(), collisions_mem.getDeviceStride(), output_row_width_bytes, output_height_reg_32, hipMemcpyDeviceToHost));
    CUDACall(hipMemcpy(storage_ptr, storage_devptr, sizeof(uint32_t)*num_reduce_storage_blocks*2, hipMemcpyDeviceToHost));
    CUDACall(hipDeviceSynchronize());

    // printf("========== Output: =============\n");
    // for(unsigned int j = 0; j < 8; j++){
    //     for(unsigned int i = 0; i < 8; i++){
    //         printf("%08X ", output_ptr_c[i + j * output_width_samples]);
    //     }
    //     printf("\n");
    // }
    // printf("================================\n");

    // printf("========== Sheet: =============\n");
    // for(unsigned int j = 0; j < 8; j++){
    //     for(unsigned int i = 0; i < 8; i++){
    //         printf("%08X ", sheet_ptr_c[i + j * sheet_width_samples]);
    //     }
    //     printf("\n");
    // }
    // printf("================================\n");

    // printf("========== Output (2264): =============\n");
    // for(unsigned int j = 0; j < 8; j++){
    //     for(unsigned int i = 2264; i < 2264+8; i++){
    //         printf("%08X ", output_ptr_c[i + j * output_width_samples]);
    //     }
    //     printf("\n");
    // }
    // printf("================================\n");

    // printf("========= Sheet (2264): ============\n");
    // for(unsigned int j = 0; j < 8; j++){
    //     for(unsigned int i = 2264; i < 2264+8; i++){
    //         printf("%08X ", sheet_ptr_c[i + j * sheet_width_samples]);
    //     }
    //     printf("\n");
    // }
    // printf("================================\n");

    printf("Reduce %u: ", num_reduce_storage_blocks);
    for(unsigned int i = 0; i < num_reduce_storage_blocks; i++){
        printf("{%u %u}, ", storage_ptr_uint32[i*2], storage_ptr_uint32[i*2+1]);
    }
    printf("\n");

    printf("Synchronized success\n");
    printf("%u\n", storage_ptr_uint32[0]);

    {
        size_t unset_memory_count = 0;
        size_t zeroed_memeory_count = 0;
        size_t correct_memory_count = 0;
        // start memory check
        for (size_t i = 0; i < local_sheet_bytes; i++)
        {
            if (sheet_ptr_c[i] == 0)
            {
                zeroed_memeory_count += 1;
            }
            else if (sheet_ptr_c[i] == ~0)
            {
                correct_memory_count += 1;
            }
            else
            {
                unset_memory_count += 1;
            }
        }

        printf("Memcheck %ldB: %ld unset, %ld zeroed, %ld correct\n",
               local_sheet_bytes, unset_memory_count,
               zeroed_memeory_count, correct_memory_count);
    }

    {
        size_t unset_memory_count = 0;
        size_t zeroed_memeory_count = 0;
        size_t correct_memory_count = 0;
        // start memory check
        for (size_t i = 0; i < local_output_bytes; i++)
        {
            if (output_ptr_c[i] == 0)
            {
                zeroed_memeory_count += 1;
            }
            else if (output_ptr_c[i] == ~0)
            {
                correct_memory_count += 1;
            }
            else
            {
                unset_memory_count += 1;
            }
        }

        printf("Memcheck %ldB: %ld unset, %ld zeroed, %ld correct\n",
               local_output_bytes, unset_memory_count,
               zeroed_memeory_count, correct_memory_count);
    }


    // debug testing
    printf("Debug memory checks\n");
    sheet_mem.Pull();
    part_mem.Pull();
    reduce_mem.Pull();
    collisions_mem.Pull();

    printf("RDC Ptr: %p\n", reduce_mem.getDevPtr_unsafe());

    printf("sheet match: %d\n", memcmp(sheet_mem.gethostPtr_unsafe(), sheet_ptr, local_sheet_bytes));
    printf("part match: %d\n", memcmp(part_mem.gethostPtr_unsafe(), c, part.getLinearDataSize()*sizeof(uint32_t)));
    printf("collisions match: %d\n", memcmp(collisions_mem.gethostPtr_unsafe(), output_ptr, local_output_bytes));
    printf("reduce match: %d\n", memcmp(reduce_mem.gethostPtr_unsafe(), storage_ptr, sizeof(uint32_t)*num_reduce_storage_blocks*2));


    printf("Local output bytes: %lu x %lu {%lu} | Maanged output bytes: %lu x %lu {??} (%lu) \n", 
        output_row_width_bytes, output_height_reg_32,
        local_output_bytes, 
        collisions_mem.getHostStride(), collisions_mem.height(),
        collisions_mem.getRawWidthBytes()
    );

    {
        char* output_ptr_m = (char*)collisions_mem.gethostPtr_unsafe();
        size_t unset_memory_count = 0;
        size_t zeroed_memeory_count = 0;
        size_t correct_memory_count = 0;
        // start memory check
        for (size_t i = 0; i < local_output_bytes; i++)
        {
            if (output_ptr_m[i] == 0)
            {
                zeroed_memeory_count += 1;
            }
            else if (output_ptr_m[i] == ~0)
            {
                correct_memory_count += 1;
            }
            else
            {
                unset_memory_count += 1;
            }
        }

        printf("Memcheck %ldB: %ld unset, %ld zeroed, %ld correct\n",
               local_output_bytes, unset_memory_count,
               zeroed_memeory_count, correct_memory_count);
    }

    free(sheet_ptr);
    free(output_ptr);
    CUDACall(hipFree(output_devptr));
    CUDACall(hipFree(sheet_devptr));
    CUDACall(hipFree(part_devptr));
}


int main(const int argc, const char * const * const argv){
    // TODO - endianness checks on entry

    std::cout << "Running: " << argv[0] << std::endl; 

    displayCUDAdeviceStats();

    // For now, assume that we want the first device always
    // Set device 0 as current
    hipSetDevice(0);


    // Position p_local;
    // Position* p_dev;
    // hipMalloc(&p_dev, sizeof(Position));
    // checkRaster<<<5, 1>>>(nullptr, nullptr, p_dev);
    // hipMemcpy(&p_local, p_dev, sizeof(Position), hipMemcpyDeviceToHost);
    // hipFree(p_dev);
    // std::cout << p_local.first << " " << p_local.second << std::endl;

    std::vector<std::string> filepaths_vector = {
        "../SampleRasters/basebot.raster",
        "../SampleRasters/blocker.raster",
        "../SampleRasters/Part7.raster",
        "../SampleRasters/vert1.raster",
        "../SampleRasters/vert2.raster",
        };

    std::vector<Raster> r_vector;

    for (const auto& fp : filepaths_vector){
        r_vector.emplace_back(readRaster(fp));
    }

    std::cout << "Loaded test rasters, startingpack." << std::endl;

    // hostPack_entry(r_vector);
    // hostPack_entry_cpu(r_vector);
    // hostPack_entry_cuda(r_vector);

    r_vector[0].print(8, 385);
    
    simple_cuda(r_vector[0]);

}